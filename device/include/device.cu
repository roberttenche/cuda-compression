#include "device.cuh"

///
/// Host - Device Implementation
///

#include "Common.cuh"
#include "device_specific.cuh"

#include <filesystem>
#include <fstream>

#include <iostream>

///
/// Huffman
///

Exit_Code Huffman::compress(std::string input_file_path, Memory_Block_Size gpu_block_size)
{
  uint8* bytes = nullptr;
  uint32 bytes_size = 0uL;

  // get file size
  bytes_size = std::filesystem::file_size(input_file_path);

  std::cout << "FILE SIZE : " << bytes_size << std::endl;

  // check that file size is big enough for at least 1 memory block
  if (bytes_size < Memory_Block_Size_to_uint64(gpu_block_size))
  {
    return exit_code = Exit_Code::FILE_TOO_SMALL;
  }

  std::ifstream input_file(input_file_path, std::ios::binary);

  // check if file can be opened
  if (!input_file.is_open())
  {
    return exit_code = Exit_Code::FILE_NOT_ACCESSIBLE;
  }



  input_file.close();

  // run huffman compression
  huffman_device_specific_compress(bytes, bytes_size, gpu_block_size);

  // free allocated memory
  hipFree(bytes);
  
  // all good
  return exit_code = Exit_Code::ALL_GOOD;
}

Exit_Code Huffman::decompress(std::string input_file_path)
{
  huffman_device_specific_decompress();

  return exit_code = Exit_Code::ALL_GOOD;
}

Exit_Code Huffman::exit_code = Exit_Code::UNTREATED_ERROR;
Memory_Block_Size Huffman::gpu_block_size = Memory_Block_Size::MEMORY_BLOCK_32MB;
Memory_Block_Size Huffman::cpu_ram_size = Memory_Block_Size::MEMORY_BLOCK_2GB;

///
/// Lempel-Ziv-Welch
///

Exit_Code Lempel_Ziv_Welch::compress(std::string input_file_path, Memory_Block_Size gpu_block_size)
{
  uint8* bytes = nullptr;
  uint32 bytes_size = 0uL;

  lzw_device_specific_compress(bytes, bytes_size, gpu_block_size);

  return exit_code = Exit_Code::ALL_GOOD;
}

Exit_Code Lempel_Ziv_Welch::decompress(std::string input_file_path)
{
  lzw_device_specific_decompress();

  return exit_code = Exit_Code::ALL_GOOD;
}

Exit_Code Lempel_Ziv_Welch::exit_code = Exit_Code::UNTREATED_ERROR;
Memory_Block_Size Lempel_Ziv_Welch::gpu_block_size = Memory_Block_Size::MEMORY_BLOCK_32MB;
Memory_Block_Size Lempel_Ziv_Welch::cpu_ram_size = Memory_Block_Size::MEMORY_BLOCK_2GB;
