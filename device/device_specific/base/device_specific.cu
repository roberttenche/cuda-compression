#include "Common.cuh"
#include "device.cuh"
#include "huffman.cuh"
#include "lempel_ziv_welch.cuh"

#include <stdio.h>

uint16 device_cuda_cores = 512u;

void huffman_device_specific_compress(void* input, void* output)
{
  //
  // Insert math here
  //

  // call algo
  huffman_compress<<<1, device_cuda_cores>>>(input, output);
  hipDeviceSynchronize();
}
void huffman_device_specific_decompress(void* input, void* output)
{
  //
  // Insert math here
  //

  // call algo
  huffman_decompress<<<1, device_cuda_cores>>>(input, output);
  hipDeviceSynchronize();
}

void lzw_device_specific_compress(void* input, void* output)
{
  //
  // Insert math here
  //

  // call algo
  lempel_ziv_welch_compress<<<1, device_cuda_cores>>>(input, output);
  hipDeviceSynchronize();
}
void lzw_device_specific_decompress(void* input, void* output)
{
  //
  // Insert math here
  //

  // call algo
  lempel_ziv_welch_decompress<<<1, device_cuda_cores>>>(input, output);
  hipDeviceSynchronize();
}
