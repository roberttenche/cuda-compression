#include "Common.cuh"
#include "device.cuh"
#include "huffman.cuh"
#include "lempel_ziv_welch.cuh"

#include <stdio.h>

static uint16 device_cuda_cores = 512u;

///
/// Huffman
///

void huffman_device_specific_compress()
{
  //
  // Insert math here
  //

  // call algo
  huffman_compress<<<1, device_cuda_cores>>>();
  hipDeviceSynchronize();
}
void huffman_device_specific_decompress()
{
  //
  // Insert math here
  //

  // call algo
  huffman_decompress<<<1, device_cuda_cores>>>();
  hipDeviceSynchronize();
}

///
/// Lempel-Ziv-Welch
///

void lzw_device_specific_compress()
{
  //
  // Insert math here
  //

  // call algo
  lempel_ziv_welch_compress<<<1, device_cuda_cores>>>();
  hipDeviceSynchronize();
}
void lzw_device_specific_decompress()
{
  //
  // Insert math here
  //

  // call algo
  lempel_ziv_welch_decompress<<<1, device_cuda_cores>>>();
  hipDeviceSynchronize();
}
