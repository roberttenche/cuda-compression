#include "hip/hip_runtime.h"
#include "lempel_ziv_welch.cuh"

#include <stdio.h>

///
/// Lempel-Ziv-Welch algorithm implementation
///

__global__ void lempel_ziv_welch_compress(void* input, void* output)
{
  printf("Lempel-Ziv-Welch compression unimplemented!!! - threadIdx:%d\n", threadIdx.x);
}
__global__ void lempel_ziv_welch_decompress(void* input, void* output)
{
  printf("Lempel-Ziv-Welch decompression unimplemented!!! - threadIdx:%d\n", threadIdx.x);
}
