#include "hip/hip_runtime.h"
#include "lempel_ziv_welch.cuh"

#include <stdio.h>

///
/// Lempel-Ziv-Welch algorithm implementation
///

///
/// PUBLIC FUNCTIONS
///
__global__ void lempel_ziv_welch_compress()
{
  printf("Lempel-Ziv-Welch compression unimplemented!!! - threadIdx:%d\n", threadIdx.x);
}
__global__ void lempel_ziv_welch_decompress()
{
  printf("Lempel-Ziv-Welch decompression unimplemented!!! - threadIdx:%d\n", threadIdx.x);
}

///
/// PRIVATE FUNCTIONS
///
